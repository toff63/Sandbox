#include <iostream>
#include <ctime>
#include <hip/hip_runtime.h>
				// Stops underlining of __global__
	// Stops underlining of threadIdx etc.

using namespace std;

__global__ void FindClosestGPU(float3* points, int* indices, int count)
{
if(count <= 1) return;

int idx = threadIdx.x + blockIdx.x * blockDim.x;
if(idx < count)
	{
	float3 thisPoint = points[idx];
	float smallestSoFar = 3.40282e38f;
	
	for(int i = 0; i < count; i++)
		{
		if(i == idx) continue;

		float dist = (thisPoint.x - points[i].x)*(thisPoint.x - points[i].x);
		dist += (thisPoint.y - points[i].y)*(thisPoint.y - points[i].y);
		dist += (thisPoint.z - points[i].z)*(thisPoint.z - points[i].z);
		
		if(dist < smallestSoFar)
			{
			smallestSoFar = dist;
			indices[idx] = i;
			}
		}
	}
}


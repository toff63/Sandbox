#include "hip/hip_runtime.h"
#include <iostream>
#include <ctime>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>				// Stops underlining of __global__
#include <>	// Stops underlining of threadIdx etc.

#include "FindClosestCPU.h"
#include "FindClosestGPU.h"

using namespace std;

int main()
{
// Number of points
const int count = 10000;

// Arrays of points
int *indexOfClosest = new int[count];
float3 *points = new float3[count];
float3* d_points;	 // GPU version
int* d_indexOfClosest;

// Create a list of random points
for(int i = 0; i < count; i++)
	{
	points[i].x = (float)((rand()%10000) - 5000);
	points[i].y = (float)((rand()%10000) - 5000);
	points[i].z = (float)((rand()%10000) - 5000);
	}

hipMalloc(&d_points, sizeof(float3) * count);
hipMemcpy(d_points, points, sizeof(float3) * count, hipMemcpyHostToDevice);
hipMalloc(&d_indexOfClosest, sizeof(int) * count);

// This variable is used to keep track of the fastest time so far
long fastest = 1000000;

// Run the algorithm 20 times
for(int q = 0; q < 20; q++)
	{
	long startTime = clock();
	
	// Run the algorithm
	//FindClosestCPU(points, indexOfClosest, count);
	
	FindClosestGPU<<<(count / 320)+1, 320>>>(d_points, d_indexOfClosest, count);
	hipMemcpy(indexOfClosest, d_indexOfClosest, sizeof(int) * count, hipMemcpyDeviceToHost);

	long finishTime = clock();
	
	cout<<q<<" "<<(finishTime - startTime)<<endl;

	// If that run was faster update the fastest time so far
	if((finishTime - startTime) < fastest)
		fastest = (finishTime - startTime);
	}

// Print out the fastest time
cout<<"Fastest time: "<<fastest<<endl;

// Print the final results to screen
cout<<"Final results:"<<endl;
for(int i = 0; i < 10; i++)
	cout<<i<<"."<<indexOfClosest[i]<<endl;
	
// Deallocate ram
delete[] indexOfClosest;
delete[] points;
hipFree(d_points);
hipFree(d_indexOfClosest);

hipDeviceReset();

return 0;
}